#include "hip/hip_runtime.h"
#include <math.h>
#include <>
#include "kernel_function.h"
#define BLOCK_SIZE 16

__global__ 
void blobFromImageKernel(const uchar* imgData, float* blob, int channels, int img_h, int img_w) 
{
    int h = blockIdx.x * blockDim.x + threadIdx.x;
    int w = blockIdx.y * blockDim.y + threadIdx.y;

    if (h < img_h && w < img_w) {
        for (int c = 0; c < channels; ++c)
        {
            blob[c * img_w * img_h + h * img_w + w] = static_cast<float>(imgData[h * img_w * channels + w * channels + c]);
        }
    }
}

__global__
void GenerateYoloProposalKernel(int* gridStrides, int gridStrideSize, float* outputSrc, 
                            float* objects, float bboxConfThresh, int numClass)
{
    // gridStrides: [8400, 3]
    // outputSrc: [bs, 8400, 4 + 1 + NumClasses]
    // objects: 1 + [8400, 7] 1: save keepFlag count 7: x0, y0, w, h, box_prob, class_idx, keepFlag
    int anchorIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int classIdx = blockIdx.y * blockDim.y + threadIdx.y;

    if (anchorIdx  >= gridStrideSize || classIdx >= numClass) 
        return;
    int x = gridStrides[anchorIdx * 3];
    int y = gridStrides[anchorIdx * 3 + 1];
    int stride = gridStrides[anchorIdx * 3 + 2];
    float* psrc = outputSrc + anchorIdx * (4 + 1 + numClass);

    float box_objecness = psrc[4];
    float box_cls_score = psrc[5 + classIdx];
    float box_prob = box_objecness * box_cls_score;
    if (box_prob > bboxConfThresh)
    {
        int index = atomicAdd(objects, 1);
        float* pobject = objects + 1 + index * 7;

        float x0 = (x + psrc[0]) * stride;
        float y0 = (y + psrc[1]) * stride;
        float w = exp(psrc[2]) * stride;
        float h = exp(psrc[3]) * stride;
        // caculate left top 
        x0 = x0 - w * 0.5f;
        y0 = y0 - h * 0.5f;
        *pobject++ = x0;
        *pobject++ = y0;
        *pobject++ = w;
        *pobject++ = h;
        *pobject++ = box_prob;
        *pobject++ = classIdx;
        *pobject++ = 1;
    }
}

__device__
float bboxIouDevice(float ax1, float ay1, float aw, float ah,
              float bx1, float by1, float bw, float bh)
{
    float inter_x1 = max(ax1, bx1);
    float inter_y1 = max(ay1, by1);
    float inter_x2 = min(ax1 + aw, bx1 + bw);
    float inter_y2 = min(ay1 + ah, by1 + bh);

    float inter_w = max(0.0f, inter_x2 - inter_x1);
    float inter_h = max(0.0f, inter_y2 - inter_y1);
    float inter_area = inter_w * inter_h;
    float box1_area = aw * ah;
    float box2_area = bw * bh;
    float union_area = box1_area + box2_area - inter_area;
    if (union_area <= 0)
        return 0;
    return inter_area / union_area;
}

__global__
void FastNMS(float* objects, float iouThresh, int objectWidth, int topK)
{
    // objects 1 + [8400, 7] 1: count, 7: x0, y0, w, h, box_prob, class_idx, keepFlag
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int count = min(int(objects[0]), topK);
    if (idx >= objects[0])
        return;
    float* pcurrent = objects + 1 + idx * objectWidth;
    if (pcurrent[6] == 0)
        return;
    for (int i = 0; i < count; ++i)
    {
        float* pitem = objects + 1 + i * objectWidth;
        if (i == idx || pitem[5] != pcurrent[5])
            continue;
        if (pitem[4] >= pcurrent[4])
        {
            if (pitem[4] == pcurrent[4] && i < idx)
                continue;
            float iou = bboxIouDevice(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                                      pitem[0], pitem[1], pitem[2], pitem[3]);
            if (iou > iouThresh)
            {
                pcurrent[6] = 0;
                return;
            }
        }
    }
}

// (x, y) -> (projX, projY)
__device__
void AffineProjectKernel(AffineMatrix* matrix, int x, int y, float* projX, float* projY)
{
    *projX = matrix->v00 * x + matrix->v01 * y + matrix->v02;
    *projY = matrix->v10 * x + matrix->v11 * y + matrix->v12;
}

__global__
void ResizeNearestPaddingKernel(uchar* src, int srcWidth, int srcHeight, float* dst, int dstWidth, int dstHeight, 
                                int widthPadding, int heightPadding, uchar paddingValue, AffineMatrix d2s)
{
    int dx = blockIdx.x * blockDim.x + threadIdx.x;  // dstWidth
    int dy = blockIdx.y * blockDim.y + threadIdx.y;  // dstHeight
    if (dx >= dstWidth || dy >= dstHeight)
        return;
    float projX = 0;
    float projY = 0;
    AffineProjectKernel(&d2s, dx, dy, &projX, &projY);
    float c0, c1, c2;
    if (dx >= dstWidth - widthPadding || dy >= dstHeight - heightPadding)
    {
        c0 = paddingValue;
        c1 = paddingValue;
        c2 = paddingValue;
    }
    else
    {
        int srcX = floor(projX + 0.5f);
        int srcY = floor(projY + 0.5f);
        
        if (srcX < 0 || srcX >= srcWidth || srcY < 0 || srcY >= srcHeight)
        {
            c0 = paddingValue;
            c1 = paddingValue;
            c2 = paddingValue;
        }
        else
        {
            c0 = static_cast<float>(src[srcY * srcWidth * 3 + srcX * 3]);
            c1 = static_cast<float>(src[srcY * srcWidth * 3 + srcX * 3 + 1]);
            c2 = static_cast<float>(src[srcY * srcWidth * 3 + srcX * 3 + 2]);
        }
    }
    float* pdst = dst + dy * dstWidth * 3 + dx * 3;
    *pdst++ = c0;
    *pdst++ = c1;
    *pdst++ = c2;
}

__global__
void ResizeBilinearPaddingKernel(uchar* src, int srcWidth, int srcHeight, float* dst, int dstWidth, int dstHeight,
                                int widthPadding, int heightPadding, uchar paddingValue, AffineMatrix d2s)
{
    int dx = blockIdx.x * blockDim.x + threadIdx.x;  // dstWidth
    int dy = blockIdx.y * blockIdx.y + threadIdx.y;  // dstHeight
    if (dx >= dstWidth || dy >= dstHeight)
        return;
    // coming soon
}
__global__
void HWC2CHWKernel(float* src, float* dst, int width, int height, int channels)
{
    int w = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    if (w >= width || h >= height)
        return;
    for (int c = 0; c < channels; ++c)
    {
        dst[c * height * width + h * width + w] = src[h * width * channels + w * channels + c];
    }
}

void FastNMSDevice(float* objects, float iouThresh, int objectWidth, int topK, const hipStream_t& stream)
{
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((topK + blockSize.x - 1) / blockSize.x);
    FastNMS<<<gridSize, blockSize, 0, stream>>>(objects, iouThresh, objectWidth, topK);
}

void GenerateYoloProposalDevice(int* gridStrides, int gridStrideSize, float* outputSrc, float* objects,
                            float bboxConfThresh, int numClass, const hipStream_t& stream)
{
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((gridStrideSize + blockSize.x - 1) / blockSize.x, (numClass + blockSize.y - 1) / blockSize.y);
    GenerateYoloProposalKernel<<<gridSize, blockSize, 0, stream>>>(gridStrides, gridStrideSize, outputSrc, objects, bboxConfThresh, numClass);
}

void ResizePaddingDevice(uchar* src, int srcWidth, int srcHeight, float* dst, int dstWidth, int dstHeight, float scale, AffineMatrix d2s, const hipStream_t& stream)
{
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((dstWidth + blockSize.x - 1) / blockSize.x, (dstHeight + blockSize.y - 1) / blockSize.y);

    int heightPadding = (dstHeight - srcHeight * scale);
    int widthPadding = (dstWidth - srcWidth * scale);
    uchar paddingValue = 114;
    ResizeNearestPaddingKernel<<<gridSize, blockSize, 0, stream>>>(src, srcWidth, srcHeight, dst, dstWidth, dstHeight, widthPadding, heightPadding, paddingValue, d2s);
}

void HWC2CHWDevice(float* src, float* dst, int width, int height, int channels, const hipStream_t& stream)
{
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    HWC2CHWKernel<<<gridSize, blockSize, 0, stream>>>(src, dst, width, height, channels);
}

void blobFromImageCuda(float* blobDev, const cv::Mat& img, const hipStream_t& stream) 
{
    int channels = img.channels();
    int img_h = img.rows;
    int img_w = img.cols;

    // Copy image data from CPU to GPU using the provided stream
    uchar* imgDataDev = nullptr;
    hipMalloc((void**)&imgDataDev, img.total() * img.channels() * sizeof(uchar));
    hipMemcpyAsync(imgDataDev, img.data, img.total() * img.channels() * sizeof(uchar), hipMemcpyHostToDevice, stream);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((img_w + blockSize.x - 1) / blockSize.x, (img_h + blockSize.y - 1) / blockSize.y);

    blobFromImageKernel<<<gridSize, blockSize, 0, stream>>>(imgDataDev, blobDev, channels, img_h, img_w);

    hipFree(imgDataDev);  // Free GPU memory used for image data
}


// int main()
// {
//     cv::Mat img = cv::imread("car.jpg");
//     int channels = img.channels();
//     int img_h = img.rows;
//     int img_w = img.cols;
//     int img_size = img.total() * img.channels();
//     uchar* imgDataDev = nullptr;
//     hipMalloc((void**)&imgDataDev, img_size * sizeof(uchar));
//     hipMemcpy(imgDataDev, img.data, img_size * sizeof(uchar), hipMemcpyHostToDevice);

//     int dst_w = 640;
//     int dst_h = 640;

//     float* dstImage = nullptr;
//     hipMalloc((void**)&dstImage, dst_w * dst_h * channels * sizeof(float));

//     float scale = std::min(dst_w / (img_w * 1.0f), dst_h / (img_h * 1.0f));

//     std::cout << scale << std::endl;

//         // 源图像上的三个点和目标图像上的对应三个点
//     cv::Point2f srcPoints[3] = {
//         cv::Point2f(0, 0),     // 左上角
//         cv::Point2f(img_w-1, 0),     // 右上角
//         cv::Point2f(0, img_h-1)      // 左下角
//     };

//     cv::Point2f dstPoints[3] = {
//         cv::Point2f(0, 0),     // 左上角
//         cv::Point2f((int) img_w * scale - 1 , 0),     // 右上角
//         cv::Point2f(0, (int) img_h* scale - 1)      // 左下角
//     };

//     cv::Mat M = cv::getAffineTransform(dstPoints, srcPoints);

//     std::cout << "Scale is: " << 1.0f / scale << std::endl;

//     std::cout << "Affine Transformation Matrix:\n" << M << std::endl;

    
//     std::cout<< "Affine Transformation Matrix:\n" << d2s.v00 << " " << d2s.v01 << " " << d2s.v02 << " " << d2s.v10 << " " << d2s.v11 << " " << d2s.v12 << std::endl;

//     AffineMatrix d2s;
//     d2s.v00 = M.at<double>(0, 0);
//     d2s.v01 = M.at<double>(0, 1);
//     d2s.v02 = M.at<double>(0, 2);
//     d2s.v10 = M.at<double>(1, 0);
//     d2s.v11 = M.at<double>(1, 1);
//     d2s.v12 = M.at<double>(1, 2);

    
//     ResizePaddingDevice(imgDataDev, img_w, img_h, dstImage, dst_w, dst_h, scale, d2s, 0);

//     float* hostData = new float[dst_w * dst_h * channels];
//     hipMemcpy(hostData, dstImage, dst_w * dst_h * channels * sizeof(float), hipMemcpyDeviceToHost);
//     cv::Mat result(dst_h, dst_w, CV_32FC(channels), hostData);

//     result.convertTo(result, CV_8UC3);
//     cv::imwrite("result.jpg", result);

//     delete[] hostData;
//     return 0;
// }